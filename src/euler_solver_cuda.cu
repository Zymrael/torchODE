#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <map>

typedef void (*solver_t)(torch::PackedTensorAccessor<float, 2>, torch::PackedTensorAccessor<float, 1>, torch::PackedTensorAccessor<float, 1>, float, int, int);
typedef void (*method_t)(double, double, double, float, int);

typedef std::string string;
typedef std::map<string, method_t> map;

__inline__ __device__ double
euler_method(double F_in, double x0_in, double g_in, float dt, int steps) {
	return x0_in + (F_in * g_in)*dt;
         //x0_in1 = (UL * g_in1 + UR * g_in2)*dt
         //x0_in2 = (LL * g_in1 + LR & g_in2)*dt
	 //F_in = 1 && g_in = UL * g_in1 + UR * g_in2;
	//return;
}

__inline__ __device__ double
rk4_method(double F_in, double x0_in, double g_in, float dt, int steps) {
	auto f1 = (F_in * g_in)*dt;

	auto c2 = dt * f1 / 2.0;
        auto f2 = (F_in * (g_in + c2)) * (dt / 2.0);

	auto c3 = dt * f2 / 2.0;
        auto f3 = (F_in * (g_in + c3)) * (dt / 2.0);

	auto c4 = dt * f3;
	auto f4 = (F_in * (g_in + c4)) * dt;

	return x0_in + (f1 + 2.0 * f2 + 2.0 * f3 + f4) / 6.0;
}


__global__ void
general_solver(method_t method, torch::PackedTensorAccessor<float, 2> F_a, torch::PackedTensorAccessor<float, 1> x0_a, torch::PackedTensorAccessor<float, 1> g_a, float dt, int steps, int W) { 
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < W){
        double x0_in = x0_a[tid];
	double g_in = g_a[tid];
        double F_in = F_a[tid][tid];

   	for(int i = 0; i < steps; i++) {
		//x0_in += (F_in * g_in)*dt;
		x0_in = euler_method(F_in, x0_in, g_in, dt, steps);
	}

        x0_a[tid] = x0_in;
    }
}

__global__ void
skew_symmetric_solver(method_t method, float UL_v, float UR_v, float LL_v, float LR_v, torch::PackedTensorAccessor<float, 1> x0_a, torch::PackedTensorAccessor<float, 1> g_a, float dt, int steps,int size) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < size/2) {
	double g_in_1 = g_a[tid];
	double g_in_2 = g_a[tid + size/2];

        double x0_in_1 = x0_a[tid];
        double x0_in_2 = x0_a[tid + size/2];

   	for(int i = 0; i < steps; i++) {
		// F_in = 1 && g_in = UL * g_in1 + UR * g_in2;
		//method(UL_v, x0_in_1, g_in_1, dt, steps);
		//method(UR_v, x0_in_2, g_in_2, dt, steps);
		//method(LL_v, x0_in_1, g_in_1, dt, steps);
		//method(LR_v, x0_in_2, g_in_2, dt, steps);
	}

        x0_a[tid] = x0_in_1;
	x0_a[tid + size/2] = x0_in_2;
    }
}

torch::Tensor solver_cuda(torch::Tensor F, torch::Tensor x0, torch::Tensor g, double dt, int steps, int W, string name){

    map methods;
    methods["Euler"] = &euler_method;
    methods["RK4"] = &rk4_method;
    method_t chosen_method = methods[name];

    auto F_a = F.packed_accessor<float,2>();
    auto x0_a = x0.packed_accessor<float,1>();
    auto g_a = g.packed_accessor<float,1>();
    //auto F_size = torch::size(F, 0);

    //auto xud = torch::chunk(x0, 2, 0);
    //auto xulr = torch::chunk(xud[0], 2, 1);
    //auto xllr = torch::chunk(xud[1], 2, 1);

/*
    auto UL = xulr[0].packed_accessor<float, 2>();
    auto UR = xulr[1].packed_accessor<float, 2>();
    auto LL = xllr[0].packed_accessor<float, 2>();
    auto LR = xllr[1].packed_accessor<float, 2>();
*/

    /*if(F_a == F_a[0][0] * torch::eye(F_size)) {
    	const int threadsPerBlock = 512;
    	const int blocks = (W + threadsPerBlock - 1) / threadsPerBlock;	

    } else if(	F_a[0] == UL 				&& F_a[F_size/2][0] == LL && 
		UR[0][0]*torch::eye(F_size/2) == UR 	&& F_a[F_size/2][F_size/2] == LR)
	// Launch UL, LL, UR, LR kernel
    	const int threadsPerBlock = 512;
    	const int blocks = (W + threadsPerBlock - 1) / threadsPerBlock;

    } else {*/
    	const int threadsPerBlock = 512; 
    	const int blocks = (W*W + threadsPerBlock - 1) / threadsPerBlock;
	general_solver<<<blocks, threadsPerBlock>>>(chosen_method, F_a, x0_a, g_a, dt, steps, W);
    //}
   return x0;
}


