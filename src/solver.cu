#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <map>

typedef void (*solver_t)(torch::PackedTensorAccessor<float, 2>, torch::PackedTensorAccessor<float, 1>, torch::PackedTensorAccessor<float, 1>, float, int, int);
typedef void (*method_t)(float, float, float, float, int);

typedef std::string string;

__device__ void 
euler_method(float F_in, float x0_in, float g_in, float dt, int steps) {
	x0_in = x0_in + (F_in * g_in) * dt;
}

__device__ void 
rk4_method(float F_in, float x0_in, float g_in, float dt, int steps) {
	auto f1 = (F_in * g_in)*dt;

	auto c2 = dt * f1 / 2.0;
        auto f2 = (F_in * (g_in + c2)) * (dt / 2.0);

	auto c3 = dt * f2 / 2.0;
        auto f3 = (F_in * (g_in + c3)) * (dt / 2.0);

	auto c4 = dt * f3;
	auto f4 = (F_in * (g_in + c4)) * dt;

	x0_in = x0_in + (f1 + 2.0 * f2 + 2.0 * f3 + f4) / 6.0;
}


__global__ void
general_solver(method_t method, torch::PackedTensorAccessor<float, 2> F_a, torch::PackedTensorAccessor<float, 1> x0_a, torch::PackedTensorAccessor<float, 1> g_a, float dt, int steps, int x0_size) { 
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < x0_size){
        auto x0_in = x0_a[tid];
	auto g_in = g_a[tid];
        auto F_in = F_a[tid][tid];

   	for(int i = 0; i < steps; i++) {
		method(F_in, x0_in, g_in, dt, steps);
	}

        x0_a[tid] = x0_in;
    }
}

__global__ void
compact_diagonal_solver(method_t method, float F_in, torch::PackedTensorAccessor<float, 1> x0_a, torch::PackedTensorAccessor<float, 1> g_a, float dt, int steps, int x0_size) { 
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < x0_size){
        auto x0_in = x0_a[tid];
	auto g_in = g_a[tid];

   	for(int i = 0; i < steps; i++) {
		method(F_in, x0_in, g_in, dt, steps);
	}

        x0_a[tid] = x0_in;
    }
}

__global__ void
compact_skew_symmetric_solver(method_t method, float UL_v, float UR_v, float LL_v, float LR_v, torch::PackedTensorAccessor<float, 1> x0_a, torch::PackedTensorAccessor<float, 1> g_a, float dt, int steps, int x0_size) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < x0_size) {
	auto g_in_1 = g_a[tid];
	auto g_in_2 = g_a[tid + x0_size/2];

        auto x0_in_1 = x0_a[tid];
        auto x0_in_2 = x0_a[tid + x0_size/2];

   	for(int i = 0; i < steps; i++) {
		method(UL_v, x0_in_1, g_in_1, dt, steps);
		method(UR_v, x0_in_2, g_in_2, dt, steps);
		method(LL_v, x0_in_1, g_in_1, dt, steps);
		method(LR_v, x0_in_2, g_in_2, dt, steps);
	}

        x0_a[tid] = x0_in_1;
	x0_a[tid + x0_size/2] = x0_in_2;
    }
}

// Declare static pointers to device functions
__device__ method_t p_euler_method = euler_method;
__device__ method_t p_rk4_method = rk4_method;

void solve(torch::Tensor F, torch::Tensor x0, torch::Tensor g, float dt, int steps, string name){

    std::map<string, method_t> h_methods;
    method_t h_euler_method;
    method_t h_rk4_method; 

    // Copy device function pointers to host side
    hipMemcpyFromSymbol(&h_euler_method, HIP_SYMBOL(p_euler_method), sizeof(method_t));
    hipMemcpyFromSymbol(&h_rk4_method, HIP_SYMBOL(p_rk4_method), sizeof(method_t));

    h_methods["Euler"] = h_euler_method;
    h_methods["RK4"] = h_rk4_method;

    method_t d_chosen_method = h_methods[name];

    auto F_a = F.packed_accessor<float,2>();
    auto x0_a = x0.packed_accessor<float,1>();
    auto g_a = g.packed_accessor<float,1>();

    auto F_size = torch::size(x0, 0);
    auto x0_size = torch::size(x0, 0);

    const int threadsPerBlock = 512; 
    const int blocks = (x0_size*x0_size + threadsPerBlock - 1) / threadsPerBlock;

    //general_solver<<<blocks, threadsPerBlock>>>(d_chosen_method, F_a, x0_a, g_a, dt, steps, x0_size);
    switch(F_size) {
	case 1:
		auto F_in = F_a[0][0];
		compact_diagonal_solver<<<blocks, threadsPerBlock>>>(d_chosen_method, F_in, x0_a, g_a, dt, steps, x0_size);
		break;
//	case 4:
//		compact_skew_symmetric_solver<<<blocks, threadsPerBlock>>>(d_chosen_method, F_a[0][0], F_a[0][1], F_a[1][0], F_a[1][1], x0_a, g_a, dt, steps, x0_size);
//		break;
	default:
		general_solver<<<blocks, threadsPerBlock>>>(d_chosen_method, F_a, x0_a, g_a, dt, steps, x0_size);
		break;
    }
}

